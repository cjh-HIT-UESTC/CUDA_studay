#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<iostream>

//cuda的一些头文件
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include<hip/hip_runtime_api.h>
#include ""

__global__ void kernal(float* a)  //线程函数 
{
	a[threadIdx.x] = 1;
}

void cjh()                         //调起函数
{
	hipSetDevice(0);              //选择设备
	float* aGpu;				//定义设备
	hipMalloc((void**)&aGpu, 16 * sizeof(float));//分配显存
	float a[16] = { 0 };                     //分配内存
	hipMemcpy(aGpu, a, 16 * sizeof(float), hipMemcpyHostToDevice);//主机到设备
	kernal << <1, 16 >> > (aGpu);								//调用线程函数
	hipMemcpy(a, aGpu, 16 * sizeof(float), hipMemcpyDeviceToHost);//设备到主机
	for (int i = 0; i < 16; ++i)		//输出结果
	{
		printf("%f", a[i]);
	}
}



