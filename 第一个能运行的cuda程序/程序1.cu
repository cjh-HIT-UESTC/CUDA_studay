#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<iostream>

//cuda的一些头文件
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include<hip/hip_runtime_api.h>
#include ""

__global__ void kernal(float* a)  //线程函数 
{
	a[threadIdx.x] = 1;
}

void cjh()//调起函数
{
	//读出设备有多少个显卡
	int gpuCount = -1;			//显卡数量初始化
	hipGetDeviceCount(&gpuCount);		//得到显卡数量 注意用了：&
	printf("gpuCount: %d\n", gpuCount);

	hipSetDevice(gpuCount - 1);	//选择最后一块设备（本机上也就是第0号）

	//hipSetDevice(0);						//选择设备
	float* aGpu;								//定义设备
	hipMalloc((void**)&aGpu, 16 * sizeof(float));//分配显存
	float a[16] = { 0 };						//分配内存
	hipMemcpy(aGpu, a, 16 * sizeof(float), hipMemcpyHostToDevice);//主机到设备
	kernal << <1, 16 >> > (aGpu);								//调用线程函数
	hipMemcpy(a, aGpu, 16 * sizeof(float), hipMemcpyDeviceToHost);//设备到主机
	for (int i = 0; i < 16; ++i)		//输出结果
	{
		printf("%f\n", a[i]);
	}
	hipFree(aGpu);		//释放显存
	hipDeviceReset(); //把设定恢复初始化 便于后续换设备

	//查设备的一些信息
	hipDeviceProp_t prop;	//定义一个结构体 这个结构体内部由一些数据构成
	hipGetDeviceProperties(&prop, 0);
	printf("maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
	printf("maxThreadsDim: %d\n", prop.maxThreadsDim);
	printf("maxGridSize: %d\n", prop.maxGridSize);
	printf("totalConstMem: %d\n", prop.totalConstMem);
	printf("clockRate: %d\n", prop.clockRate);
	printf("integrated: %d\n", prop.integrated);

	//自动选择最优的设备
	int deviceId;		//先定义设备编号
	hipChooseDevice(&deviceId, &prop);	//传入定义的编号 和 查到的设备信息结构体 自动返回选到的设备编号到定义的数字里去
	printf("deviceId: %d\n", deviceId);

	//因为本机只有一个设备 如果要分配两个设备会报错 捕捉它的错误 并给出如何分配两个设备
	//int deviceList[2] = { 0,1 };
	//hipSetValidDevices(deviceList, 1);




	
}



